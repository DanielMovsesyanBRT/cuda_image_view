#include "hip/hip_runtime.h"
/**
 *
 *
 *
 */
#include "debayer.hpp"
#include "image_processor.hpp"

namespace brt {
namespace jupiter {


__constant__ size_t color_map[image::eNumTypes][image::NumColors] =
{
  //            Blue = 0, Green = 1, Red = 2,  Alpha = 3, Bayer = 4
  /*eNone = 0*/  {  0,       0,          0,        0,          0     },
  /*eBayer = 1*/ {  0,       0,          0,        0,          0     },
  /*eRGB = 2*/   {  0,       1,          2,        0,          0     },
  /*eBGR = 3*/   {  2,       1,          0,        0,          0     },
  /*eRGBA= 4*/   {  0,       1,          2,        3,          0     },
  /*eBGRA= 5*/   {  2,       1,          0,        3,          0     },
};

size_t type_size[image::eNumTypes] =
{
  /*eNone = 0*/  1,
  /*eBayer = 1*/ 1,
  /*eRGB = 2*/   3,
  /*eBGR = 3*/   3,
  /*eRGBA= 4*/   4,
  /*eBGRA= 5*/   4,
};

__constant__ double             _Xn = (0.950456);
__constant__ double             _Zn = (1.088754);


class MemPtr
{
public:
  __device__ MemPtr(hipPitchedPtr mem,int element_size,int xoffset = 2,int yoffset = 2)
  : _mem(mem), _el_size(element_size), _xo(xoffset), _yo(yoffset)
  {   }

protected:
  __device__ uint8_t* ptr(int x,int y)
  {
    return ((uint8_t*)_mem.ptr) + ((x + _xo) * _el_size + (y + _yo) * _mem.pitch);
  }

private:
  hipPitchedPtr                  _mem;
  int                             _el_size;
  int                             _xo,_yo;
};


/*
 * \\class Bayer
 *
 * created on: Feb 12, 2020, 3:08:28 PM
 * author daniel
 *
 */
class Bayer : public MemPtr
{
public:
  __device__ Bayer(hipPitchedPtr mem,int xoffset = 2,int yoffset = 2)
  : MemPtr(mem,sizeof(uint16_t), xoffset, yoffset) { }

  __device__ uint16_t& operator()(int x,int y)
  {
    return *(uint16_t*)ptr(x,y);
  }
};


/*
 * \\class Pix
 *
 * created on: Feb 12, 2020, 3:08:36 PM
 * author daniel
 *
 */
class Pix : public MemPtr
{
public:
  __device__ Pix(hipPitchedPtr mem,image::PixelType type,int typesize,int xoffset = 2,int yoffset = 2)
  : MemPtr(mem,typesize * sizeof(uint16_t), xoffset, yoffset)
  , _type(type)
  { }

  __device__ uint16_t& operator()(int x,int y,image::Color cl)
  {
    return *(uint16_t*)(ptr(x,y) + color_map[_type][cl] * sizeof(uint16_t));
  }

private:
  image::PixelType                _type;
};


/*
 * \\class LAB
 *
 * created on: Feb 12, 2020, 3:10:01 PM
 * author daniel
 *
 */
class LAB
{
public:
  __device__ LAB() : _L(0.0),_a(0.0),_b(0.0) {}

  __device__ void   fromRGB(uint16_t r,uint16_t g,uint16_t b)
  {
    double X,Y,Z;

    // Matrix multiplication
    X = (0.412453 * static_cast<double>(r)  +
         0.357580 * static_cast<double>(g)  +
         0.180423 * static_cast<double>(b)) / _Xn;

    Y = (0.212671 * static_cast<double>(r) +
         0.715160 * static_cast<double>(g) +
         0.072169 * static_cast<double>(b));

    Z = (0.019334 * static_cast<double>(r) +
         0.119193 * static_cast<double>(g) +
         0.950227 * static_cast<double>(b)) / _Zn;

    auto adjust = [](double value)->double
    {
      return (value > 0.00856) ? pow(value,0.33333333333) : (7.787 * value + 0.1379310);
    };

    _L = (Y > 0.00856) ? (116.0 * pow(Y,0.33333333333) - 16.0) : 903.3 * Y;
    _a = 500.0 * (adjust(X) - adjust(Y));
    _b = 200.0 * (adjust(Y) - adjust(Z));
  }

  __device__ double               L() const { return _L; }
  __device__ double               a() const { return _a; }
  __device__ double               b() const { return _b; }

private:
  double                          _L;
  double                          _a;
  double                          _b;
};


/*
 * \\class Lab
 *
 * created on: Feb 12, 2020, 3:17:40 PM
 * author daniel
 *
 */
class Lab : public MemPtr
{
public:
  __device__ Lab(hipPitchedPtr mem,int xoffset = 2,int yoffset = 2)
  : MemPtr(mem,sizeof(LAB), xoffset, yoffset)
  { }

  __device__ LAB& operator()(int x,int y)
  {
    return *(LAB*) ptr(x,y);
  }
};


/*
 * \\fn void green_interpolate
 *
 * created on: Feb 11, 2020, 4:25:08 PM
 * author daniel
 *
 */
__global__ void green_interpolate(image::PixelType type, int typesize,
                                  hipPitchedPtr raw_image,
                                  hipPitchedPtr horiz_image,
                                  hipPitchedPtr vertical_image)
{
  int origx = ((blockIdx.x * blockDim.x) + threadIdx.x) << 1;
  int origy = ((blockIdx.y * blockDim.y) + threadIdx.y) << 1;

  Bayer raw(raw_image);
  Pix hr(horiz_image,type,typesize);
  Pix vr(vertical_image,type,typesize);

  auto limit = [](int x,int a,int b)->int
  {
    int result = max(x,min(a,b));
    return min(result,max(a,b));
  };

  // C R
  // B C
  // (0,0) -> Clear
  int x = origx, y = origy;
  vr(x,y,image::Green) = hr(x,y,image::Green) = raw(x,y);
  vr(x,y,image::Alpha) = hr(x,y,image::Alpha) = (uint16_t)-1;

  ////////////////////////////////////////////////
  // (1,0) -> Red
  x = origx + 1;
  y = origy;

  int value = (((raw(x-1,y) + raw(x,y) + raw(x+1,y)) * 2) - raw(x - 2,y) - raw(x + 2,y)) >> 2;
  hr(x,y,image::Green) = limit(value,raw(x - 1,y),raw(x + 1,y));

  value = (((raw(x,y-1) + raw(x,y) + raw(x,y+1)) * 2) - raw(x,y-2) - raw(x,y+2)) >> 2;
  vr(x,y,image::Green) = limit(value,raw(x,y-1),raw(x,y+1));

  vr(x,y,image::Alpha) = hr(x,y,image::Alpha) = (uint16_t)-1;

  ////////////////////////////////////////////////
  // (0,1) -> Blue
  x = origx;
  y = origy + 1;

  value = (((raw(x-1,y) + raw(x,y) + raw(x+1,y)) * 2) - raw(x - 2,y) - raw(x + 2,y)) >> 2;
  hr(x,y,image::Green) = limit(value,raw(x - 1,y),raw(x + 1,y));

  value = (((raw(x,y-1) + raw(x,y) + raw(x,y+1)) * 2) - raw(x,y-2) - raw(x,y+2)) >> 2;
  vr(x,y,image::Green) = limit(value,raw(x,y-1),raw(x,y+1));

  vr(x,y,image::Alpha) = hr(x,y,image::Alpha) = (uint16_t)-1;

  ////////////////////////////////////////////////
  // (1,1) -> Clear
  x = origx + 1;
  y = origy + 1;

  vr(x,y,image::Green) = hr(x,y,image::Green) = raw(x,y);
  vr(x,y,image::Alpha) = hr(x,y,image::Alpha) = (uint16_t)-1;
}


/*
 * \\fn void blue_red_interpolate
 *
 * created on: Feb 12, 2020, 11:35:37 AM
 * author daniel
 *
 */
__global__ void blue_red_interpolate(image::PixelType type, int typesize,
                                      hipPitchedPtr raw_image,
                                      hipPitchedPtr horiz_image,
                                      hipPitchedPtr vertical_image,
                                      hipPitchedPtr hlab,
                                      hipPitchedPtr vlab)
{
  int origx = ((blockIdx.x * blockDim.x) + threadIdx.x) << 1;
  int origy = ((blockIdx.y * blockDim.y) + threadIdx.y) << 1;

  Bayer raw(raw_image);
  Pix hr(horiz_image,type,typesize);
  Pix vr(vertical_image,type,typesize);

  Lab hl(hlab);
  Lab vl(vlab);


  auto limit = [](int x,int a,int b)->int
  {
    int result = max(x,min(a,b));
    return min(result,max(a,b));
  };

  // C R
  // B C

  ////////////////////////////////////////////////
  // (0,0) -> ClearRead
  int x = origx, y = origy;

  // Horizontal
  int value = hr(x,y,image::Green) + ((raw(x-1,y) - hr(x-1,y,image::Green) + raw(x+1,y) - hr(x+1,y,image::Green)) >> 1);
  hr(x,y,image::Red) = limit(value, 0, ((1 << 16) - 1));

  value = hr(x,y,image::Green) + ((raw(x,y-1) - hr(x,y-1,image::Green) + raw(x,y+1) - hr(x,y+1,image::Green)) >> 1);
  hr(x,y,image::Blue) = limit(value, 0, ((1 << 16) - 1));
  hl(x,y).fromRGB(hr(x,y,image::Red),hr(x,y,image::Green),hr(x,y,image::Blue));

  // Vertical
  value = vr(x,y,image::Green) + ((raw(x-1,y) - vr(x-1,y,image::Green) + raw(x+1,y) - vr(x+1,y,image::Green)) >> 1);
  vr(x,y,image::Red) = limit(value, 0, ((1 << 16) - 1));

  value = vr(x,y,image::Green) + ((raw(x,y-1) - vr(x,y-1,image::Green) + raw(x,y+1) - vr(x,y+1,image::Green)) >> 1);
  vr(x,y,image::Blue) = limit(value, 0, ((1 << 16) - 1));
  vl(x,y).fromRGB(vr(x,y,image::Red),vr(x,y,image::Green),vr(x,y,image::Blue));

  ////////////////////////////////////////////////
  // (1,0) -> Red
  x = origx + 1;
  y = origy;

  hr(x,y,image::Red) = vr(x,y,image::Red) = raw(x,y);

  value = hr(x,y,image::Green) + ((raw(x-1,y-1) - hr(x-1,y-1,image::Green) +
                                   raw(x-1,y+1) - hr(x-1,y+1,image::Green) +
                                   raw(x+1,y-1) - hr(x+1,y-1,image::Green) +
                                   raw(x+1,y+1) - hr(x+1,y+1,image::Green)) >> 2);
  hr(x,y,image::Blue) = limit(value, 0, ((1 << 16) - 1));
  hl(x,y).fromRGB(hr(x,y,image::Red),hr(x,y,image::Green),hr(x,y,image::Blue));

  value = vr(x,y,image::Green) + ((raw(x-1,y-1) - vr(x-1,y-1,image::Green) +
                                   raw(x-1,y+1) - vr(x-1,y+1,image::Green) +
                                   raw(x+1,y-1) - vr(x+1,y-1,image::Green) +
                                   raw(x+1,y+1) - vr(x+1,y+1,image::Green)) >> 2);
  vr(x,y,image::Blue) = limit(value, 0, ((1 << 16) - 1));
  vl(x,y).fromRGB(vr(x,y,image::Red),vr(x,y,image::Green),vr(x,y,image::Blue));

  ////////////////////////////////////////////////
  // (0,1) -> Blue
  x = origx;
  y = origy + 1;

  hr(x,y,image::Blue) = vr(x,y,image::Blue) = raw(x,y);

  value = hr(x,y,image::Green) + ((raw(x-1,y-1) - hr(x-1,y-1,image::Green) +
                                   raw(x-1,y+1) - hr(x-1,y+1,image::Green) +
                                   raw(x+1,y-1) - hr(x+1,y-1,image::Green) +
                                   raw(x+1,y+1) - hr(x+1,y+1,image::Green)) >> 2);
  hr(x,y,image::Red) = limit(value, 0, ((1 << 16) - 1));
  hl(x,y).fromRGB(hr(x,y,image::Red),hr(x,y,image::Green),hr(x,y,image::Blue));

  value = vr(x,y,image::Green) + ((raw(x-1,y-1) - vr(x-1,y-1,image::Green) +
                                   raw(x-1,y+1) - vr(x-1,y+1,image::Green) +
                                   raw(x+1,y-1) - vr(x+1,y-1,image::Green) +
                                   raw(x+1,y+1) - vr(x+1,y+1,image::Green)) >> 2);
  vr(x,y,image::Red) = limit(value, 0, ((1 << 16) - 1));
  vl(x,y).fromRGB(vr(x,y,image::Red),vr(x,y,image::Green),vr(x,y,image::Blue));

  ////////////////////////////////////////////////
  // (1,1) -> ClearBlue
  x = origx + 1;
  y = origy + 1;

  // Horizontal
  value = hr(x,y,image::Green) + ((raw(x-1,y) - hr(x-1,y,image::Green) + raw(x+1,y) - hr(x+1,y,image::Green)) >> 1);
  hr(x,y,image::Blue) = limit(value, 0, ((1 << 16) - 1));

  value = hr(x,y,image::Green) + ((raw(x,y-1) - hr(x,y-1,image::Green) + raw(x,y+1) - hr(x,y+1,image::Green)) >> 1);
  hr(x,y,image::Red) = limit(value, 0, ((1 << 16) - 1));
  hl(x,y).fromRGB(hr(x,y,image::Red),hr(x,y,image::Green),hr(x,y,image::Blue));

  // Vertical
  value = vr(x,y,image::Green) + ((raw(x-1,y) - vr(x-1,y,image::Green) + raw(x+1,y) - vr(x+1,y,image::Green)) >> 1);
  vr(x,y,image::Blue) = limit(value, 0, ((1 << 16) - 1));

  value = vr(x,y,image::Green) + ((raw(x,y-1) - vr(x,y-1,image::Green) + raw(x,y+1) - vr(x,y+1,image::Green)) >> 1);
  vr(x,y,image::Red) = limit(value, 0, ((1 << 16) - 1));
  vl(x,y).fromRGB(vr(x,y,image::Red),vr(x,y,image::Green),vr(x,y,image::Blue));
}

/*
 * \\fn void misguidance_color_artifacts
 *
 * created on: Feb 12, 2020, 3:21:22 PM
 * author daniel
 *
 */
__global__ void misguidance_color_artifacts(image::PixelType type, int typesize,
                                            hipPitchedPtr result_image,
                                            hipPitchedPtr horiz_image,
                                            hipPitchedPtr vertical_image,
                                            hipPitchedPtr hlab,
                                            hipPitchedPtr vlab,
                                            uint32_t* histogram,uint32_t histogram_size,
                                            uint32_t* small_histogram,uint32_t small_histogram_size)
{
  int hist_size_bits = ((sizeof(unsigned int) * 8) - 1 -  __clz(histogram_size));
  int small_hist_size_bits = ((sizeof(unsigned int) * 8) - 1 -  __clz(small_histogram_size));

  int x = ((blockIdx.x * blockDim.x) + threadIdx.x);
  int y = ((blockIdx.y * blockDim.y) + threadIdx.y);

  Pix rst(result_image,type,typesize,0,0);
  Pix hr(horiz_image,type,typesize);
  Pix vr(vertical_image,type,typesize);

  Lab hl(hlab);
  Lab vl(vlab);

  auto sqr=[](double a)->double { return a*a; };

  double lv[2],lh[2],cv[2],ch[2];
  int hh = 0,hv = 0;

  lh[0] = fabs(hl(x,y).L() - hl(x-1,y).L());
  lh[1] = fabs(hl(x,y).L() - hl(x+1,y).L());

  lv[0] = fabs(vl(x,y).L() - vl(x,y-1).L());
  lv[1] = fabs(vl(x,y).L() - vl(x,y+1).L());

  ch[0] = sqr(hl(x,y).a() - hl(x-1,y).a()) +
          sqr(hl(x,y).b() - hl(x-1,y).b());

  ch[1] = sqr(hl(x,y).a() - hl(x+1,y).a()) +
          sqr(hl(x,y).b() - hl(x+1,y).b());

  cv[0] = sqr(vl(x,y).a() - vl(x,y-1).a()) +
          sqr(vl(x,y).b() - vl(x,y-1).b());

  cv[1] = sqr(vl(x,y).a() - vl(x,y+1).a()) +
          sqr(vl(x,y).b() - vl(x,y+1).b());

  double eps_l = min(max(lh[0],lh[1]),max(lv[0],lv[1]));
  double eps_c = min(max(ch[0],ch[1]),max(cv[0],cv[1]));

  for (size_t index = 0; index < 2; index++)
  {
    if ((lh[index] <= eps_l) && (ch[index] <= eps_c))
      hh++;

    if ((lv[index] <= eps_l) && (cv[index] <= eps_c))
      hv++;
  }

  uint32_t r = 0,g = 0,b = 0;
  if (hh > hv)
  {
    r = rst(x,y,image::Red) = hr(x,y,image::Red);
    g = rst(x,y,image::Green) = hr(x,y,image::Green);
    b = rst(x,y,image::Blue) = hr(x,y,image::Blue);
    rst(x,y,image::Alpha) = hr(x,y,image::Alpha);
  }
  else if (hv > hh)
  {
    r = rst(x,y,image::Red) = vr(x,y,image::Red);
    g = rst(x,y,image::Green) = vr(x,y,image::Green);
    b = rst(x,y,image::Blue) = vr(x,y,image::Blue);
    rst(x,y,image::Alpha) = vr(x,y,image::Alpha);
  }
  else //if (hv == hh)
  {
    r = rst(x,y,image::Red) = (hr(x,y,image::Red) + vr(x,y,image::Red)) >> 1;
    g = rst(x,y,image::Green) = (hr(x,y,image::Green) + vr(x,y,image::Green)) >> 1;
    b = rst(x,y,image::Blue) = (hr(x,y,image::Blue) + vr(x,y,image::Blue)) >> 1;
    rst(x,y,image::Alpha) = (uint16_t)-1;
  }

  uint32_t brightness = ((r+r+r+b+g+g+g+g) >> 3) * small_histogram_size >> small_hist_size_bits;
  atomicAdd(&small_histogram[brightness & ((1 << small_hist_size_bits) - 1)], 1);

  brightness = ((r+r+r+b+g+g+g+g) >> 3) * histogram_size >> hist_size_bits;
  atomicAdd(&histogram[brightness & ((1 << hist_size_bits) - 1)], 1);
}


/*
 * \\fn void cudaMax
 *
 * created on: Nov 22, 2019
 * author: daniel
 *
 */
__global__ void cudaMax(uint32_t *org, uint32_t *max)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  max[tid] = org[tid];

  auto step_size = 1;
  int number_of_threads = gridDim.x * blockDim.x;

  __syncthreads();

  while (number_of_threads > 0)
  {
    if (tid < number_of_threads)
    {
      const auto fst = tid * step_size * 2;
      const auto snd = fst + step_size;

      max[fst] = (max[fst] < max[snd]) ? max[snd] : max[fst];
    }

    step_size <<= 1;
    number_of_threads >>= 1;
  }
}


/*
 * \\class CudaImage
 *
 * created on: Feb 11, 2020
 *
 */
class Debayer_Impl
{
friend Debayer;
public:
  Debayer_Impl();
  virtual ~Debayer_Impl();

          bool                    init(size_t width,size_t height,image::PixelType,size_t small_hits_size);
          image::RawRGBPtr        ahd(image::RawRGBPtr);
private:
          void                    free();

private:
  hipPitchedPtr                  _raw_image;
  hipPitchedPtr                  _horiz_image;
  hipPitchedPtr                  _vertical_image;
  hipPitchedPtr                  _result_image;
  hipPitchedPtr                  _hlab;
  hipPitchedPtr                  _vlab;

  void*                           _histogram;
  void*                           _histogram_max;
  size_t                          _histogram_size;

  void*                           _small_histogram;
  size_t                          _small_histogram_size;

  size_t                          _width;
  size_t                          _height;
  image::PixelType                _type;

  int                             _thx,_thy;
  int                             _blkx,_blky;
};

/*
 * \\fn Constructor Debayer_Impl::Debayer_Impl
 *
 * created on: Feb 11, 2020, 3:05:08 PM
 * author daniel
 *
 */
Debayer_Impl::Debayer_Impl()
: _raw_image{nullptr,0,0,0}
, _horiz_image{nullptr,0,0,0}
, _vertical_image{nullptr,0,0,0}
, _result_image{nullptr,0,0,0}
, _hlab{nullptr,0,0,0}
, _vlab{nullptr,0,0,0}
, _histogram(nullptr)
, _histogram_max(nullptr)
, _histogram_size(0)
, _small_histogram(nullptr)
, _small_histogram_size(0)
, _width(0)
, _height(0)
, _type(image::eNone)
, _thx(0)
, _thy(0)
, _blkx(0)
, _blky(0)
{
}

/*
 * \\fn Destructor Debayer_Impl::~Debayer_Impl
 *
 * created on: Feb 11, 2020, 3:05:58 PM
 * author daniel
 *
 */
Debayer_Impl::~Debayer_Impl()
{
  free();
}

/*
 * \\fn bool Debayer_Impl::init
 *
 * created on: Feb 11, 2020, 3:41:45 PM
 * author daniel
 *
 */
bool Debayer_Impl::init(size_t width,size_t height,image::PixelType type,size_t small_hits_size)
{
  if ((_width == width) && (_height == height) && (type_size[type] == type_size[_type]))
    return true;

  free();

  _width = width;
  _height = height;
  _type = type;
  _small_histogram_size = small_hits_size;

  // Raw image with extra 2 pixels at each end....
  hipError_t err = hipMalloc3D(&_raw_image, make_hipExtent((_width + 4) * sizeof(uint16_t) * type_size[image::eBayer],_height + 4, 1));
  if (err != hipSuccess)
    return false;

  err = hipMemset3D(_raw_image, 0, make_hipExtent((_width + 4) * sizeof(uint16_t) * type_size[image::eBayer],_height + 4, 1));
  if (err != hipSuccess)
    return false;
  //////////////////////////

  err = hipMalloc3D(&_horiz_image, make_hipExtent((_width + 4) * type_size[_type] * sizeof(uint16_t),_height + 4, 1));
  if (err != hipSuccess)
    return false;

  err = hipMemset3D(_horiz_image, 0, make_hipExtent((_width + 4) * type_size[_type] * sizeof(uint16_t),_height + 4, 1));
  if (err != hipSuccess)
    return false;
  //////////////////////////

  err = hipMalloc3D(&_vertical_image, make_hipExtent((_width + 4) * type_size[_type] * sizeof(uint16_t),_height + 4, 1));
  if (err != hipSuccess)
    return false;

  err = hipMemset3D(_vertical_image, 0, make_hipExtent((_width + 4) * type_size[_type] * sizeof(uint16_t),_height + 4, 1));
  if (err != hipSuccess)
    return false;
  //////////////////////////

  err = hipMalloc3D(&_result_image, make_hipExtent(_width * type_size[_type] * sizeof(uint16_t),_height, 1));
  if (err != hipSuccess)
    return false;

  err = hipMemset3D(_result_image, 0, make_hipExtent(_width * type_size[_type] * sizeof(uint16_t),_height, 1));
  if (err != hipSuccess)
    return false;
  //////////////////////////

  err = hipMalloc3D(&_hlab, make_hipExtent((_width + 4) * sizeof(double) * 3,_height + 4, 1));
  if (err != hipSuccess)
    return false;

  err = hipMemset3D(_hlab, 0, make_hipExtent((_width + 4) * sizeof(double) * 3,_height + 4, 1));
  if (err != hipSuccess)
    return false;
  //////////////////////////

  err = hipMalloc3D(&_vlab, make_hipExtent((_width + 4) * sizeof(double) * 3,_height + 4, 1));
  if (err != hipSuccess)
    return false;

  err = hipMemset3D(_vlab, 0, make_hipExtent((_width + 4) * sizeof(double) * 3,_height + 4, 1));
  if (err != hipSuccess)
    return false;
  //////////////////////////

  _histogram_size = (1 << 16);
  err = hipMalloc(&_histogram,_histogram_size * sizeof(uint32_t));
  if (err != hipSuccess)
    return false;

  err = hipMalloc(&_histogram_max,_histogram_size * sizeof(uint32_t));
  if (err != hipSuccess)
    return false;


  err = hipMalloc(&_small_histogram,_small_histogram_size * sizeof(uint32_t));
  if (err != hipSuccess)
    return false;


  _thx = std::min(DEFAULT_NUMBER_OF_THREADS, (1 << __builtin_ctz(_width)));
  if (_thx == 0)
    _thx = 1;

  _thy = std::min(DEFAULT_NUMBER_OF_THREADS, (1 << __builtin_ctz(_height)));
  if (_thy == 0)
    _thy = 1;

  _blkx = _width / _thx;
  if ((_width % _thx) != 0)
    _blkx++;

  _blky = _height / _thy;
  if ((_height % _thy) != 0)
    _blky++;

  return true;
}

/*
 * \\fn void Debayer_Impl::free
 *
 * created on: Feb 11, 2020, 3:54:09 PM
 * author daniel
 *
 */
void Debayer_Impl::free()
{
  if (_raw_image.pitch != 0)
    hipFree(_raw_image.ptr);

  if (_horiz_image.pitch != 0)
    hipFree(_horiz_image.ptr);

  if (_vertical_image.pitch != 0)
    hipFree(_vertical_image.ptr);

  if (_result_image.pitch != 0)
    hipFree(_result_image.ptr);

  if (_hlab.pitch != 0)
    hipFree(_hlab.ptr);

  if (_vlab.pitch != 0)
    hipFree(_vlab.ptr);

  hipFree(_histogram);
  hipFree(_histogram_max);
  hipFree(_small_histogram);
}


/*
 * \\fn image::RawRGBPtr Debayer_Impl::ahd
 *
 * created on: Feb 11, 2020, 3:14:04 PM
 * author daniel
 *
 */
image::RawRGBPtr Debayer_Impl::ahd(image::RawRGBPtr img)
{
  if (!img)
    return image::RawRGBPtr();

  hipMemcpy3DParms mcp = {0};
  mcp.srcPtr.ptr  = img->bytes();
  mcp.srcPtr.pitch = img->width() * sizeof(uint16_t);
  mcp.srcPtr.xsize = img->width() * sizeof(uint16_t);
  mcp.srcPtr.ysize = img->height();

  mcp.dstPtr.ptr = _raw_image.ptr;
  mcp.dstPtr.pitch = _raw_image.pitch;
  mcp.dstPtr.xsize = _raw_image.xsize;
  mcp.dstPtr.ysize = _raw_image.ysize;

  mcp.dstPos.x     = 2 * sizeof(uint16_t);
  mcp.dstPos.y     = 2;
  mcp.dstPos.z     = 0;

  mcp.extent.width  = img->width()* sizeof(uint16_t);
  mcp.extent.height = img->height();
  mcp.extent.depth  = 1;

  mcp.kind = hipMemcpyHostToDevice;
  assert(hipMemcpy3D(&mcp) == hipSuccess);

  //
  dim3 threads(_thx >> 1,_thy >> 1);
  dim3 blocks(_blkx, _blky);

  green_interpolate<<<blocks,threads>>>(_type, type_size[_type],
                                        _raw_image,
                                        _horiz_image,
                                        _vertical_image);
  hipDeviceSynchronize();

  blue_red_interpolate<<<blocks,threads>>>(_type, type_size[_type],
                                        _raw_image,
                                        _horiz_image,
                                        _vertical_image,
                                        _hlab,
                                        _vlab);
  hipDeviceSynchronize();

  dim3 threads2(_thx,_thy);

  hipMemset(_histogram,0,_histogram_size * sizeof(uint32_t));
  hipMemset(_small_histogram,0,_small_histogram_size * sizeof(uint32_t));

  misguidance_color_artifacts<<<blocks,threads2>>>(_type, type_size[_type],
                                        _result_image,
                                        _horiz_image,
                                        _vertical_image,
                                        _hlab,
                                        _vlab,
                                        (uint32_t*)_histogram, _histogram_size,
                                        (uint32_t*)_small_histogram,_small_histogram_size);
  hipDeviceSynchronize();


  int thx = 64;
  while (_histogram_size < thx)
    thx >>= 1;

//  assert(hipMemcpy(_histogram_max, _histogram, _histogram_size * sizeof(uint32_t),hipMemcpyDeviceToDevice) == hipSuccess);
  cudaMax<<<_histogram_size / thx, thx>>>((uint32_t*)_histogram, (uint32_t*)_histogram_max);

  image::RawRGBPtr result(new image::RawRGB(img->width(), img->height(), img->depth(), _type));
  memset(&mcp,0,sizeof(mcp));

  mcp.dstPtr.ptr  = result->bytes();
  mcp.dstPtr.pitch = result->width() * type_size[_type] * sizeof(uint16_t);
  mcp.dstPtr.xsize = result->width() * type_size[_type] * sizeof(uint16_t);
  mcp.dstPtr.ysize = result->height();

  mcp.srcPtr.ptr = _result_image.ptr;
  mcp.srcPtr.pitch = _result_image.pitch;
  mcp.srcPtr.xsize = _result_image.xsize;
  mcp.srcPtr.ysize = _result_image.ysize;

  mcp.extent.width  = result->width() * type_size[_type] * sizeof(uint16_t);
  mcp.extent.height = result->height();
  mcp.extent.depth  = 1;

  mcp.kind = hipMemcpyDeviceToHost;

  assert(hipMemcpy3D(&mcp) == hipSuccess);
  return result;
}


/*
 * \\fn Constructor Debayer::Debayer
 *
 * created on: Feb 12, 2020, 4:25:18 PM
 * author daniel
 *
 */
Debayer::Debayer()
: _impl(new Debayer_Impl())
{

}

/*
 * \\fn Destructor Debayer::~Debayer
 *
 * created on: Feb 12, 2020, 4:26:01 PM
 * author daniel
 *
 */
Debayer::~Debayer()
{
  delete _impl;
}

/*
 * \\fn bool Debayer::init
 *
 * created on: Feb 12, 2020, 4:26:33 PM
 * author daniel
 *
 */
bool Debayer::init(size_t width,size_t height,image::PixelType type,size_t small_hits_size)
{
  return _impl->init(width,height,type,small_hits_size);
}

/*
 * \\fn image::RawRGBPtr Debayer::debayer
 *
 * created on: Feb 12, 2020, 4:27:24 PM
 * author daniel
 *
 */
image::RawRGBPtr Debayer::debayer(image::RawRGBPtr img)
{
  if (!img)
    return image::RawRGBPtr();

  return _impl->ahd(img);
}

/*
 * \\fn Debayer::get_histogram
 *
 * created on: Feb 13, 2020
 * author: daniel
 *
 */
bool Debayer::get_histogram(image::HistPtr& histogram)
{
  if (!histogram)
    histogram.reset(new image::Histogram);

  if (histogram->_histogram.size() != _impl->_histogram_size)
    histogram->_histogram.resize(_impl->_histogram_size);

  assert(hipMemcpy(histogram->_histogram.data(), _impl->_histogram, _impl->_histogram_size * sizeof(uint32_t),hipMemcpyDeviceToHost) == hipSuccess);

  if (histogram->_small_hist.size() != _impl->_small_histogram_size)
    histogram->_small_hist.resize(_impl->_small_histogram_size);

  assert(hipMemcpy(histogram->_small_hist.data(), _impl->_small_histogram, _impl->_small_histogram_size * sizeof(uint32_t),hipMemcpyDeviceToHost) == hipSuccess);
  assert(hipMemcpy(&histogram->_max_value, _impl->_histogram_max, sizeof(uint32_t),hipMemcpyDeviceToHost) == hipSuccess);

  return true;
}

}
// jupiter
} // brt
