#include "hip/hip_runtime.h"
/*
 * debayer_bilin.cu
 *
 *  Created on: Feb 19, 2020
 *      Author: daniel
 */


#include "debayer_bilin.hpp"
#include "cuda_2d_mem.hpp"
#include "cuda_mem.hpp"

#include <hip/hip_runtime_api.h>

#define BITS_PER_PIXEL                      (1 << 16)
#define SMALL_HIST_SIZE                     (9)

namespace brt
{
namespace jupiter
{

// Debayer an CRBC bayered image using bilinear interpolation
// and output debayered image in RGBRGBRGB (or BGRBGRBGR) in its
// original resolution.
__global__ void runCudaDebayer(const uint16_t* bayeredImg,
                               uint16_t* debayeredImg,
                               size_t width,
                               size_t height,
                               uint32_t* histogram,
                               uint32_t hist_size,
                               int hist_size_bits)
{
  // The bayered image must have the following format (when expanded to 2D):
  //
  // C R C R C R
  // B C B C B C
  // C R C R C R
  // B C B C B C
  // C R C R C R
  // B C B C B C
  //
  // where upper left corner (i.e. the first element in the array is C,
  // and the second element in the array is R).
  //
  // Other format might work, but requires rewriting this kernel
  // Otherwise the color will be messed up.
  //
  // Also, each pixel in the original bayered image but be 12 bits
  // which stored in a 16 bit uint16_t structure.
  //
  // We will treat C channel as G channel in this kernel, because during image capture,
  // we have already set the proper gain for R and B.
  //
  // (x, y) is the coordinate how we will inspect the bayered pattern
  // Note that x and y are only even numbers, meaning that
  // in every kernel, we will perform bilinear interpolation for four pixels.
  // Therefore, for image size of 1920 * 1208, this kernel is called 960 * 604 times

  int x = 2 * ((blockIdx.x * blockDim.x) + threadIdx.x);
  int y = 2 * ((blockIdx.y * blockDim.y) + threadIdx.y);

  uint32_t b, g, r;
  uint32_t brightness;

  /* Upper left: C */
  if (x == 0 && y == 0)
  {
    g = bayeredImg[y * width + x];
    r = bayeredImg[y * width + (x + 1)];
    b = bayeredImg[(y + 1) * width + x];
  }
  else if (x == 0)
  {
    g = bayeredImg[y * width + x];
    r = bayeredImg[y * width + (x + 1)];
    b = (bayeredImg[(y - 1) * width + x] + bayeredImg[(y + 1) * width + x]) / 2;
  }
  else if (y == 0)
  {
    g = bayeredImg[y * width + x];
    r = (bayeredImg[y * width + (x - 1)] + bayeredImg[y * width + (x + 1)]) / 2;
    b = bayeredImg[(y + 1) * width + x];
  }
  else
  {
    g = bayeredImg[y * width + x];
    r = (bayeredImg[y * width + (x - 1)] + bayeredImg[y * width + (x + 1)]) / 2;
    b = (bayeredImg[(y - 1) * width + x] + bayeredImg[(y + 1) * width + x]) / 2;
  }

  debayeredImg[4 * (y * width + x)] = r;
  debayeredImg[4 * (y * width + x) + 1] = g;
  debayeredImg[4 * (y * width + x) + 2] = b;
  debayeredImg[4 * (y * width + x) + 3] = (uint16_t) -1;

  if (histogram != nullptr)
  {
    brightness = ((uint32_t) (r + r + r + b + g + g + g + g) >> 3) * hist_size
        >> hist_size_bits;
    atomicAdd(&histogram[brightness & ((1 << hist_size_bits) - 1)], 1);

    //
//      if ((brightness == ((1 << hist_size_bits) - 1)) && overexpose)
//      {
//        debayeredImg[3 * (y * width + x)] = 0xFFFF;
//        debayeredImg[3 * (y * width + x) + 1] = 0;
//        debayeredImg[3 * (y * width + x) + 2] = 0;
//      }
  }

  /* Upper right: R */
  if (x == width - 2 && y == 0)
  {
    r = bayeredImg[y * width + (x + 1)];
    g = (bayeredImg[y * width + x] + bayeredImg[(y + 1) * width + (x + 1)]) / 2;
    b = bayeredImg[(y + 1) * width + x];
  }
  else if (y == 0)
  {
    r = bayeredImg[y * width + (x + 1)];
    g = (bayeredImg[y * width + x] + bayeredImg[y * width + (x + 2)]
        + bayeredImg[(y + 1) * width + (x + 1)]) / 3;

    b =
        (bayeredImg[(y + 1) * width + x] + bayeredImg[(y + 1) * width + (x + 2)])
            / 2;
  }
  else if (x == width - 2)
  {
    r = bayeredImg[y * width + (x + 1)];
    g = (bayeredImg[y * width + x] + bayeredImg[(y - 1) * width + (x + 1)]
        + bayeredImg[(y + 1) * width + (x + 1)]) / 3;

    b = (bayeredImg[(y - 1) * width + x] + bayeredImg[(y + 1) * width + x]) / 2;
  }
  else
  {
    r = bayeredImg[y * width + (x + 1)];
    g = (bayeredImg[y * width + x] + bayeredImg[y * width + (x + 2)]
        + bayeredImg[(y - 1) * width + (x + 1)]
        + bayeredImg[(y + 1) * width + (x + 1)]) / 4;

    b = (bayeredImg[(y - 1) * width + x] + bayeredImg[(y - 1) * width + (x + 2)]
        + bayeredImg[(y + 1) * width + x]
        + bayeredImg[(y + 1) * width + (x + 2)]) / 4;
  }

  debayeredImg[4 * (y * width + (x + 1))] = r;
  debayeredImg[4 * (y * width + (x + 1)) + 1] = g;
  debayeredImg[4 * (y * width + (x + 1)) + 2] = b;
  debayeredImg[4 * (y * width + (x + 1)) + 3] = (uint16_t) -1;

  if (histogram != nullptr)
  {
    brightness = ((uint32_t) (r + r + r + b + g + g + g + g) >> 3) * hist_size
        >> hist_size_bits;
    atomicAdd(&histogram[brightness & ((1 << hist_size_bits) - 1)], 1);

    //
//      if ((brightness == ((1 << hist_size_bits) - 1)) && overexpose)
//      {
//        debayeredImg[3 * (y * width + (x+1))] = 0xFFFF;
//        debayeredImg[3 * (y * width + (x+1)) + 1] = 0;
//        debayeredImg[3 * (y * width + (x+1)) + 2] = 0;
//      }
  }

  /* Lower left: B */
  if (x == 0 && y == height - 2)
  {
    b = bayeredImg[(y + 1) * width + x];
    r = bayeredImg[y * width + (x + 1)];
    g = (bayeredImg[y * width + x] + bayeredImg[(y + 1) * width + (x + 1)]) / 2;
  }
  else if (x == 0)
  {
    b = bayeredImg[(y + 1) * width + x];
    r =
        (bayeredImg[y * width + (x + 1)] + bayeredImg[(y + 2) * width + (x + 1)])
            / 2;
    g = (bayeredImg[y * width + x] + bayeredImg[(y + 1) * width + (x + 1)]
        + bayeredImg[(y + 2) * width + x]) / 3;
  }
  else if (y == height - 2)
  {
    b = bayeredImg[(y + 1) * width + x];
    r = (bayeredImg[y * width + (x - 1)] + bayeredImg[y * width + (x + 1)]) / 2;
    g = (bayeredImg[y * width + x] + bayeredImg[(y + 1) * width + (x + 1)]
        + bayeredImg[(y + 1) * width + (x - 1)]) / 3;
  }
  else
  {
    b = bayeredImg[(y + 1) * width + x];
    r = (bayeredImg[y * width + (x - 1)] + bayeredImg[y * width + (x + 1)]
        + bayeredImg[(y + 2) * width + (x - 1)]
        + bayeredImg[(y + 2) * width + (x + 1)]) / 4;

    g = (bayeredImg[y * width + x] + bayeredImg[(y + 1) * width + (x + 1)]
        + bayeredImg[(y + 2) * width + x]
        + bayeredImg[(y + 1) * width + (x - 1)]) / 4;
  }

  debayeredImg[4 * ((y + 1) * width + x)] = r;
  debayeredImg[4 * ((y + 1) * width + x) + 1] = g;
  debayeredImg[4 * ((y + 1) * width + x) + 2] = b;
  debayeredImg[4 * ((y + 1) * width + x) + 3] = (uint16_t) -1;

  if (histogram != nullptr)
  {
    brightness = ((uint32_t) (r + r + r + b + g + g + g + g) >> 3) * hist_size
        >> hist_size_bits;
    atomicAdd(&histogram[brightness & ((1 << hist_size_bits) - 1)], 1);

//    //
//    if ((brightness == ((1 << hist_size_bits) - 1)) && overexpose)
//    {
//      debayeredImg[3 * ((y + 1) * width + x)] = 0xFFFF;
//      debayeredImg[3 * ((y + 1) * width + x) + 1] = 0;
//      debayeredImg[3 * ((y + 1) * width + x) + 2] = 0;
//    }
  }

  /* Lower right: C */
  if (x == width - 2 && y == height - 2)
  {
    g = bayeredImg[(y + 1) * width + (x + 1)];
    r = bayeredImg[y * width + (x + 1)];
    b = bayeredImg[(y + 1) * width + x];
  }
  else if (x == width - 2)
  {
    g = bayeredImg[(y + 1) * width + (x + 1)];
    r = (bayeredImg[y * width + (x + 1)] + bayeredImg[(y + 2) * width + (x + 1)]) / 2;
    b = bayeredImg[(y + 1) * width + x];
  }
  else if (y == height - 2)
  {
    g = bayeredImg[(y + 1) * width + (x + 1)];
    r = bayeredImg[y * width + (x + 1)];
    b = (bayeredImg[(y + 1) * width + x] + bayeredImg[(y + 1) * width + (x + 2)]) / 2;
  }
  else
  {
    g = bayeredImg[(y + 1) * width + (x + 1)];
    r = (bayeredImg[y * width + (x + 1)] + bayeredImg[(y + 2) * width + (x + 1)]) / 2;
    b = (bayeredImg[(y + 1) * width + x] + bayeredImg[(y + 1) * width + (x + 2)]) / 2;
  }

  debayeredImg[4 * ((y + 1) * width + (x + 1))] = r;
  debayeredImg[4 * ((y + 1) * width + (x + 1)) + 1] = g;
  debayeredImg[4 * ((y + 1) * width + (x + 1)) + 2] = b;
  debayeredImg[4 * ((y + 1) * width + (x + 1)) + 3] = (uint16_t) -1;

  if (histogram != nullptr)
  {
    brightness = ((uint32_t) (r + r + r + b + g + g + g + g) >> 3) * hist_size
        >> hist_size_bits;
    atomicAdd(&histogram[brightness & ((1 << hist_size_bits) - 1)], 1);

//    //
//    if ((brightness == ((1 << hist_size_bits) - 1)) && overexpose)
//    {
//      debayeredImg[3 * ((y + 1) * width + (x + 1))] = 0xFFFF;
//      debayeredImg[3 * ((y + 1) * width + (x + 1)) + 1] = 0;
//      debayeredImg[3 * ((y + 1) * width + (x + 1)) + 2] = 0;
//    }
  }
}


/*
 * \\fn __global__ void reduceHistogram
 *
 * created on: Nov 26, 2019
 * author: daniel
 *
 */
__global__ void reduceHistogram(uint32_t* big_hist, uint32_t big_size,
                                uint32_t* small_hist, uint32_t small_size)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int sm_index = tid * small_size / big_size;
  atomicAdd(&small_hist[sm_index], big_hist[tid]);
}

/*
 * \\fn void cudaMax_bl
 *
 * created on: Nov 22, 2019
 * author: daniel
 *
 */
__global__ void cudaMax_bl(uint32_t* org,uint32_t* max)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  max[tid] = org[tid];

  auto step_size = 1;
  int number_of_threads = gridDim.x * blockDim.x;

  while (number_of_threads > 0)
  {
    if (tid < number_of_threads)
    {
      const auto fst = tid * step_size * 2;
      const auto snd = fst + step_size;

      max[fst] = (max[fst] < max[snd]) ? max[snd] : max[fst];
    }

    step_size <<= 1;
    number_of_threads >>= 1;
  }
}

/*
 * \\class Debayer_Bilinear_Impl
 *
 * created on: Feb 19, 2020
 *
 */
class Debayer_Bilinear_Impl
{
friend Debayer_Bilinear;
public:
  Debayer_Bilinear_Impl()
  : _width(0), _height(0)
  , _thx(0),_thy(0)
  , _blkx(0),_blky(0)
  { }

  virtual ~Debayer_Bilinear_Impl() {}

          image::RawRGBPtr        ahd(image::RawRGBPtr img);

private:
  CudaPtr<uint16_t>               _img_buffer;
  CudaPtr<uint16_t>               _img_debayer_buffer;
  CudaPtr<uint32_t>               _histogram;
  CudaPtr<uint32_t>               _histogram_max;
  CudaPtr<uint32_t>               _small_histogram;

  uint16_t                        _width;
  uint16_t                        _height;
  int                             _thx,_thy;
  int                             _blkx,_blky;
};




/*
 * \\fn constructor Debayer_Bilinear::Debayer_Bilinear
 *
 * created on: Feb 14, 2020
 * author: daniel
 *
 */
Debayer_Bilinear::Debayer_Bilinear()
: _width(0)
, _height(0)
{
  _impl = new Debayer_Bilinear_Impl();
}

/*
 * \\fn Debayer_Bilinear::~Debayer_Bilinear
 *
 * created on: Feb 14, 2020
 * author: daniel
 *
 */
Debayer_Bilinear::~Debayer_Bilinear()
{
  delete _impl;
}

/*
 * \\fn image::RawRGBPtr Debayer_Bilinear::ahd
 *
 * created on: Feb 19, 2020
 * author: daniel
 *
 */
image::RawRGBPtr Debayer_Bilinear::ahd(image::RawRGBPtr raw)
{
  return _impl->ahd(raw);
}

/*
 * \\fn Debayer_Bilinear::consume
 *
 * created on: Feb 19, 2020
 * author: daniel
 *
 */
void Debayer_Bilinear::consume(image::ImageBox box)
{

}

/*
 * \\fn image::RawRGBPtr Debayer_Bilinear_Impl::ahd
 *
 * created on: Feb 19, 2020
 * author: daniel
 *
 */
image::RawRGBPtr Debayer_Bilinear_Impl::ahd(image::RawRGBPtr raw)
{
  if (!raw)
    return image::RawRGBPtr();

  size_t img_size = raw->width() * raw->height();
  if (!_img_buffer.put((uint16_t*)raw->bytes(), img_size))
    return image::RawRGBPtr();

  size_t debayer_img_size = img_size * 4; /* RGBA*/

  if (!_img_debayer_buffer || (_img_debayer_buffer.size() != debayer_img_size))
    _img_debayer_buffer = CudaPtr<uint16_t>(debayer_img_size);

  if (!_img_debayer_buffer)
    return image::RawRGBPtr();

  if (!_histogram || (_histogram.size() != BITS_PER_PIXEL))
    _histogram = CudaPtr<uint32_t>(BITS_PER_PIXEL);

  if (!_histogram_max || (_histogram_max.size() != BITS_PER_PIXEL))
    _histogram_max = CudaPtr<uint32_t>(BITS_PER_PIXEL);

  if (!_small_histogram || (_small_histogram.size() != SMALL_HIST_SIZE))
    _small_histogram = CudaPtr<uint32_t>(SMALL_HIST_SIZE);

  // Check, whether image dimensions have changed
  if ((_width != raw->width()) || (_height != raw->height()))
  {
    _width = raw->width();
    _height = raw->height();

    _thx = std::min(DEFAULT_NUMBER_OF_THREADS, (1 << __builtin_ctz(_width >> 1)));
    if (_thx == 0)
      _thx = 1;

    _thy = std::min(DEFAULT_NUMBER_OF_THREADS, (1 << __builtin_ctz(_height >> 1)));
    if (_thy == 0)
      _thy = 1;

    _blkx = (_width >> 1) / _thx;
    if (((_width >> 1) % _thx) != 0)
      _blkx++;

    _blky = (_height >> 1) / _thy;
    if (((_height >> 1) % _thy) != 0)
      _blky++;
  }

  _histogram.fill(0);
  _histogram_max.fill(0);
  _small_histogram.fill(0);

  int hist_size_bits = ((sizeof(unsigned long) * 8) - 1 - __builtin_clzl(_histogram.size()));

  hipProfilerStart();

  dim3 threads(_thx,_thy);
  dim3 blocks(_blkx, _blky);

  runCudaDebayer<<<blocks,threads>>>(_img_buffer.ptr(),
                                     _img_debayer_buffer.ptr(),
                                     _width, _height,
                                     _histogram.ptr(),
                                     _histogram.size(),
                                     hist_size_bits);

  int thx = 64;
  while (_histogram.size() < thx)
    thx >>= 1;

  cudaMax_bl<<<_histogram.size() / thx, thx>>>(_histogram.ptr(), _histogram_max.ptr());

  reduceHistogram<<<_histogram.size() / thx, thx>>>(_histogram.ptr(),
                                                    _histogram.size(),
                                                    _small_histogram.ptr(),
                                                    _small_histogram.size());

  hipProfilerStop();

  image::RawRGBPtr result(new image::RawRGB(raw->width(), raw->height(), raw->depth(), image::eRGBA));
  _img_debayer_buffer.get((uint16_t*)result->bytes(), debayer_img_size);

  image::HistPtr  full_hist(new image::Histogram);
  full_hist->_histogram.resize(_histogram.size());
  full_hist->_small_hist.resize(_small_histogram.size());

  _histogram.get((uint32_t*)full_hist->_histogram.data(), _histogram.size());
  _small_histogram.get(full_hist->_small_hist.data(), _small_histogram.size());
  _histogram_max.get(&full_hist->_max_value, 1);

  result->set_histogram(full_hist);

  return result;
}


} // jupiter
} // brt
